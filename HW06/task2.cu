#include <iostream>
#include <hip/hip_runtime.h>
#include "stencil.cuh"

#include <random>           // To generate random numbers
using namespace std;


int main(int argc, char *argv[])
{
    // command line arguments
    unsigned int n = std::stoi(argv[1]);                        // n-dimension
    unsigned int R = std::stoi(argv[2]);
    unsigned int threads_per_block = std::stoi(argv[3]);        // threads per block
    
    // declarations for calculating time
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float ms;                                                   // time in ms

    // random number generation
    std::random_device entropy_source;
    std::mt19937_64 generator(entropy_source());
    std::uniform_real_distribution<float> dist1(-1.0, 1.0);
    std::uniform_real_distribution<float> dist2(-1.0, 1.0);


    // host arrays 
    float *image= (float*)malloc(n*(sizeof(float)));
    float *output= (float*)malloc(n*(sizeof(float)));
    float *mask= (float*)malloc((2*R+1)*(sizeof(float)));

    for (size_t i = 0; i < n; i++)
    {
        image[i] = dist1(generator);
    }

    for (size_t i = 0; i < (2*R+1); i++)
    {
        mask[i] = dist2(generator);
    }

    // device arrays
    float *d_image, *d_mask, *d_output;

    // allocate memory on the device (GPU)
    hipMalloc((void **)&d_image, n * sizeof(float));
    hipMalloc((void **)&d_mask, (2*R+1) * sizeof(float));
    hipMalloc((void **)&d_output, n * sizeof(float));

    // copy data into device blocks
    hipMemcpy(d_image, image, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, (2*R+1) * sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(d_output, output, n * sizeof(float), hipMemcpyHostToDevice);


    // #############################################
    hipEventRecord(start);
    stencil(d_image, d_mask, d_output, n, R, threads_per_block);
    hipEventRecord(stop);
    // #############################################
    
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);




    // Durations are converted to milliseconds already thanks to std::chrono::duration_cast
    //std::cout << "Hello World"<<std::endl;
    //std::cout << C[0] << std::endl;
    std::cout << output[n - 1] << std::endl;
    std::cout << ms << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // deallocate memory
    hipFree(d_image);
    hipFree(d_mask);
    hipFree(d_output);
    free(image);
    free(mask);
    free(output);


    return 0;
    
}
