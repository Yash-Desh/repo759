#include <hip/hip_runtime.h>
#include <cstdio>
#include <random>
#include "vscale.cuh"
#include <iostream>

int main(int argc, char *argv[])
{
    const int NUM_THREADS_PER_BLOCK = 512;
    hipEvent_t start;
    hipEvent_t stop;
    float ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int N = std::stoi(argv[1]);
    std::random_device entropy_source;
    std::mt19937_64 generator(entropy_source()); 
    std::uniform_real_distribution<float> dist1(-10.0,10.0);
    std::uniform_real_distribution<float> dist2(0.0,1.0);
    
    float*a = (float*)malloc(N*(sizeof(float)));
    float*b = (float*)malloc(N*(sizeof(float)));
    float*d_a,*d_b;


    for(int i = 0; i < N; i++)
    {
        a[i] = dist1(generator);
        b[i] = dist2(generator);
    }

    hipMalloc((void**)&d_a,sizeof(float) * N);
    hipMalloc((void**)&d_b,sizeof(float) * N);
    
    hipMemcpy(d_a,a,sizeof(float)*N,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,sizeof(float)*N,hipMemcpyHostToDevice);
    
    hipEventRecord(start);
    vscale<<<(N+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(d_a,d_b,N);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    hipMemcpy(b,d_b,sizeof(float)*N,hipMemcpyDeviceToHost);
    hipEventElapsedTime(&ms, start, stop);
  
    std::cout<<N;
    std::cout<<std::endl; 
    std::cout<<ms;
    std::cout << std::endl;
    std::cout<<b[0];
    std::cout << std::endl;
    std::cout<<b[N-1];
    std::cout << std::endl;
    std::cout << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    free(a);
    free(b);
    
}