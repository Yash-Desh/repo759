#include "hip/hip_runtime.h"
#include "vscale.cuh"
#include <hip/hip_runtime.h>

__global__ void vscale(const float *a, float *b, unsigned int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        b[index] = a[index] * b[index];
}