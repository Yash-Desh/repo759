#include <hip/hip_runtime.h>
#include "reduce.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <random> // To generate random numbers

// using namespace std;

int main(int argc, char *argv[]) {
    int N = std::stoi(argv[1]); // Array dimension
    unsigned int threads_per_block = std::stoi(argv[2]); // Block dimension

    // Time calculation variables
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float ms = 0;

    // Random number generation
    std::random_device entropy_source;
    std::mt19937_64 generator(entropy_source());
    std::uniform_real_distribution<float> dist(-1.0, 1.0);

    int num_blocks = (N + threads_per_block - 1) / threads_per_block;

    float *h_input = (float *)malloc(N * sizeof(float));
    // float *h_output = (float *)malloc(num_blocks * sizeof(float));

    for (int i = 0; i < N; i++) {
       //  h_input[i] = dist(generator);
	    h_input[i] = 1;
    }

    float *d_input;
    float *d_output;
    hipMalloc((void **)&d_input, N * sizeof(float));
    hipMalloc((void **)&d_output, num_blocks * sizeof(float));

    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);

    // #############################################
    hipEventRecord(start);
    reduce(&d_input, &d_output, N, threads_per_block);
    hipEventRecord(stop);
    // #############################################

    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(h_input, d_input, num_blocks * sizeof(float), hipMemcpyDeviceToHost);

    // Print results
    std::cout << "Reduction result: " << h_input[0] << std::endl;
    std::cout << "Elapsed time (ms): " << ms << std::endl;

    // Cleanup
    free(h_input);
    // free(h_output);
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}

// #include <hip/hip_runtime.h>
// #include "reduce.cuh"
// #include <iostream>
// #include <random>

// int main(int argc, char *argv[]) {
//     if (argc != 3) {
//         std::cerr << "Usage: " << argv[0] << " <N> <threads_per_block>" << std::endl;
//         return 1;
//     }

//     unsigned int N = std::stoi(argv[1]);
//     unsigned int threads_per_block = std::stoi(argv[2]);

//     unsigned int num_blocks = (N + threads_per_block - 1) / threads_per_block;

//     // Random number generation
//     std::random_device rd;
//     std::mt19937 gen(rd());
//     std::uniform_real_distribution<float> dis(-1.0f, 1.0f);

//     float *h_input = (float *)malloc(N * sizeof(float));
//     for (unsigned int i = 0; i < N; ++i) {
//         h_input[i] = dis(gen);
//     }

//     float *d_input, *d_output;
//     hipMalloc((void **)&d_input, N * sizeof(float));
//     hipMalloc((void **)&d_output, num_blocks * sizeof(float));

//     hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);

//     hipEvent_t start, stop;
//     hipEventCreate(&start);
//     hipEventCreate(&stop);

//     // Start timing
//     hipEventRecord(start);

//     reduce(&d_input, &d_output, N, threads_per_block);

//     // Stop timing
//     hipEventRecord(stop);
//     hipEventSynchronize(stop);

//     float ms = 0;
//     hipEventElapsedTime(&ms, start, stop);

//     float result;
//     hipMemcpy(&result, d_input, sizeof(float), hipMemcpyDeviceToHost);

//     std::cout << "Reduction result: " << result << std::endl;
//     std::cout << "Time (ms): " << ms << std::endl;

//     // Free memory
//     free(h_input);
//     hipFree(d_input);
//     hipFree(d_output);

//     return 0;
// }
