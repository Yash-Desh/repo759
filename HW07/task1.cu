#include <iostream>
#include <hip/hip_runtime.h>
#include "matmul.cuh"
#include <hip/hip_runtime.h>
#include <random>           // To generate random numbers
using namespace std;


int main(int argc, char *argv[])
{
    // command line arguments
    unsigned int n = std::stoi(argv[1]);                // arrray dimension
    unsigned int block_dim = std::stoi(argv[2]);        // block dimension
    
    // declarations for calculating time
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float ms_1=0;   
    float ms_2=0;
    float ms_3=0;                                                // time in ms

    // random number generation
    std::random_device entropy_source;
    std::mt19937_64 generator(entropy_source());

    std::uniform_int_distribution<int> dist1(-1, 1);
    std::uniform_real_distribution<float> dist2(-1.0, 1.0);
    std::uniform_real_distribution<double> dist3(-1.0, 1.0);


    // -------------------- host arrays --------------------
    int *h_A_1= (int*)malloc(n*n*(sizeof(int)));
    int *h_B_1= (int*)malloc(n*n*(sizeof(int)));
    int *h_C_1= (int*)malloc(n*n*(sizeof(int)));

    float *h_A_2= (float*)malloc(n*n*(sizeof(float)));
    float *h_B_2= (float*)malloc(n*n*(sizeof(float)));
    float *h_C_2= (float*)malloc(n*n*(sizeof(float)));

    double *h_A_3= (double*)malloc(n*n*(sizeof(double)));
    double *h_B_3= (double*)malloc(n*n*(sizeof(double)));
    double *h_C_3= (double*)malloc(n*n*(sizeof(double)));



    for (size_t i = 0; i < n*n; i++)
    {
        h_A_1[i] = dist1(generator);
        h_B_1[i] = dist1(generator);
        h_C_1[i] = 0;

        h_A_2[i] = dist2(generator);
        h_B_2[i] = dist2(generator);
        h_C_2[i] = 0;

        h_A_3[i] = dist3(generator);
        h_B_3[i] = dist3(generator);
        h_C_3[i] = 0;
    }

    // -------------------- device arrays --------------------
    int *d_A_1, *d_B_1, *d_C_1;
    float *d_A_2, *d_B_2, *d_C_2;
    double *d_A_3, *d_B_3, *d_C_3;

    // -------------------- allocate memory on the device (GPU) --------------------
    hipMalloc((void **)&d_A_1, n * n *sizeof(int));
    hipMalloc((void **)&d_B_1, n * n* sizeof(int));
    hipMalloc((void **)&d_C_1, n * n* sizeof(int));

    hipMalloc((void **)&d_A_2, n * n *sizeof(float));
    hipMalloc((void **)&d_B_2, n * n* sizeof(float));
    hipMalloc((void **)&d_C_2, n * n* sizeof(float));


    hipMalloc((void **)&d_A_3, n * n *sizeof(double));
    hipMalloc((void **)&d_B_3, n * n* sizeof(double));
    hipMalloc((void **)&d_C_3, n * n* sizeof(double));


    // -------------------- copy data into device blocks --------------------

    hipMemcpy(d_A_1, h_A_1, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B_1, h_B_1, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_C_1, h_C_1, n * n * sizeof(int), hipMemcpyHostToDevice);


    hipMemcpy(d_A_2, h_A_2, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B_2, h_B_2, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C_2, h_C_2, n * n * sizeof(float), hipMemcpyHostToDevice);


    hipMemcpy(d_A_3, h_A_3, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B_3, h_B_3, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C_3, h_C_3, n * n * sizeof(double), hipMemcpyHostToDevice);

    // ----------------------------------- int -----------------------------------

    // #############################################
    hipEventRecord(start);
    matmul_1(d_A_1, d_B_1, d_C_1, n, block_dim);
    hipEventRecord(stop);
    // #############################################
    
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms_1, start, stop);

    hipMemcpy(h_C_1, d_C_1, n * n * sizeof(float), hipMemcpyDeviceToHost);

    cout<<"hello world"<<endl;
    std::cout << h_C_1[0] << std::endl;
    std::cout << h_C_1[n*n - 1] << std::endl;
    std::cout << ms_1 << std::endl;


    // ----------------------------------- float -----------------------------------

    // #############################################
    hipEventRecord(start);
    matmul_2(d_A_2, d_B_2, d_C_2, n, block_dim);
    hipEventRecord(stop);
    // #############################################
    
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms_2, start, stop);

    hipMemcpy(h_C_2, d_C_2, n * n * sizeof(float), hipMemcpyDeviceToHost);

    cout<<"hello world"<<endl;
    std::cout << h_C_2[0] << std::endl;
    std::cout << h_C_2[n*n - 1] << std::endl;
    std::cout << ms_2 << std::endl;


    // ----------------------------------- double -----------------------------------

    // #############################################
    hipEventRecord(start);
    matmul_3(d_A_3, d_B_3, d_C_3, n, block_dim);
    hipEventRecord(stop);
    // #############################################
    
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms_3, start, stop);

    hipMemcpy(h_C_3, d_C_3, n * n * sizeof(float), hipMemcpyDeviceToHost);

    // cout<<"hello world"<<endl;
    std::cout << h_C_3[0] << std::endl;
    std::cout << h_C_3[n*n - 1] << std::endl;
    std::cout << ms_3 << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // deallocate memory
    // hipFree(d_A);
    // hipFree(d_B);
    // hipFree(d_C);
    // free(A);
    // free(B);
    // free(C);


    return 0;
    
}
