#include "hip/hip_runtime.h"
#include "reduce.cuh"
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void reduce_kernel(float *g_idata, float *g_odata, unsigned int n)
{
    extern __shared__ float shared_mem[];
    float* sdata = shared_mem;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    if(i+blockDim.x < n)
   	 sdata[threadIdx.x] = g_idata[i] + g_idata[i+blockDim.x];
    else if(i < n)
	 sdata[threadIdx.x] = g_idata[i];
    else
	 sdata[threadIdx.x] = 0;
    __syncthreads();
    for(unsigned int s = blockDim.x/2; s > 0; s>>=1)
    {
	
        if(threadIdx.x < s)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }
    if(threadIdx.x == 0) g_odata[blockIdx.x] = sdata[0];
}


__host__ void reduce(float **input, float **output, unsigned int N,unsigned int threads_per_block)
{
    while(N > 1)
    {
    unsigned int num_blocks = (N + threads_per_block - 1)/threads_per_block;

    reduce_kernel<<<num_blocks,threads_per_block,threads_per_block*sizeof(float)>>>(*input,*output,N);

    hipDeviceSynchronize();

    *input = *output;

    N = num_blocks;
	    float temp;
	    hipMemcpy(&temp,*input,sizeof(float),hipMemcpyDeviceToHost);
    //printf("Input[0] = %f\n",temp);
    }
    *input = *output;

}
